#include <stdio.h>
#include <hip/hip_runtime.h>

#define Width 1920
#define Height 2520
#define iterations 100


__global__ void convolution_kernel(unsigned char* in_device_buffer,unsigned char* out_device_buffer);
void swap_images(unsigned char **in_image,unsigned char **out_image);


int main (){
	
	unsigned char *in_image ;
	unsigned char *in_device_buffer;
	unsigned char *out_image ;
	unsigned char *out_device_buffer ;
	int grid, i;
	grid = Width* Height;
	in_image =(unsigned char*)malloc(grid);
	out_image =(unsigned char*)malloc(grid);
	
	
	FILE *fp;
	fp = fopen("images/grey_X1.raw","rb");
	if (fp == NULL){
		printf("Empty file ... Exiting");
		exit(1);
	}else {
		fread(in_image,grid,1,fp);
		fclose(fp);
	}
	 
	
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	
	dim3 threadNum(32,32);
	dim3 blockNum(Width/((threadNum.x)-2),Height/((threadNum.y)-2));
	
	
	
	
	
	
	hipMalloc(&in_device_buffer,grid);
	hipMalloc(&out_device_buffer,grid);
	hipMemcpy(in_device_buffer,in_image,grid,hipMemcpyHostToDevice);
	
	hipEventRecord(start, 0);
	for (i = 0 ; i < iterations; i++){
		convolution_kernel <<<blockNum,threadNum>>> (in_device_buffer, out_device_buffer);
		swap_images(&in_device_buffer,&out_device_buffer);	
	}
	hipEventRecord(stop, 0);


	hipMemcpy(out_image, out_device_buffer, grid, hipMemcpyDeviceToHost);
	hipFree(out_device_buffer);
	hipFree(in_device_buffer);
	
	
	hipEventSynchronize(stop);
	
	fp = fopen("out.raw","w+");
	fwrite(out_image,grid,1,fp);
	fclose(fp);
	
return 0;

}


void swap_images(unsigned char **in_image,unsigned char **out_image)
{
	unsigned char* temp = *in_image;
	*in_image = *out_image;
	*out_image = temp;
}

__global__ void convolution_kernel(unsigned char* A, unsigned char* B)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	
	int x = i-2*blockIdx.x-1;
	int y = j-2*blockIdx.y-1;
	
	__shared__ unsigned char As[32][32];
	
	//Copy from global memory to shared memory	
		
		if (x<0) {
			x=0;
		} else if (x==Width) {
			x=Width-1;
		}
		if (y<0) {
			y=0;
		} else if (y == Height) {
			y = Height-1;
		}
		As[threadIdx.x][threadIdx.y] = A[Width*y + x];
		
		__syncthreads();
	
	// Computations
	
		if (threadIdx.x!=0 && threadIdx.x!=31 && threadIdx.y!=0 && threadIdx.y!=31) {
			B[Width*y + x] =     (As[threadIdx.x-1][threadIdx.y-1]  +
										As[threadIdx.x  ][threadIdx.y-1] * 2 +
										As[threadIdx.x+1][threadIdx.y-1]  +
										As[threadIdx.x-1][threadIdx.y  ] *2 +
										As[threadIdx.x  ][threadIdx.y  ] *4 +
										As[threadIdx.x+1][threadIdx.y  ] * 2 +
										As[threadIdx.x-1][threadIdx.y+1] * 1 +
										As[threadIdx.x  ][threadIdx.y+1] * 2 +
										As[threadIdx.x+1][threadIdx.y+1] * 1)/16;
		}
}
